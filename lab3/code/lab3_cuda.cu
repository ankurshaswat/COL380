#include "hip/hip_runtime.h"
#include "lab3_cuda.h"

#define BLOCKSIZE 32
#define MAXBLOCKS 65535
#define JACOBI_TOLERANCE 0.001

__device__ inline int INDEX(int i1, int i2, int l1, int l2) {
  return i1 * l2 + i2;
}

__global__ void printMat(double *mat, int n1, int n2) {
  printf("\n");
  for (int i = 0; i < n1; i++) {
    for (int j = 0; j < n2; j++) {
      printf("%f ", mat[INDEX(i, j, n1, n2)]);
    }
    printf("\n");
  }
  printf("\n");
}

__global__ void printVec(double *vec, int n1) {
  printf("\n");
  for (int i = 0; i < n1; i++) {
    printf("%f ", vec[i]);
  }
  printf("\n");
  printf("\n");
}

__device__ void printVecDev(double *vec, int n1) {
  printf("\n");
  for (int i = 0; i < n1; i++) {
    printf("%f ", vec[i]);
  }
  printf("\n");
  printf("\n");
}

__global__ void printVec(bool *vec, int n1) {
  printf("\n");
  for (int i = 0; i < n1; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
  printf("\n");
}

__global__ void printVec(int *vec, int n1) {
  printf("\n");
  for (int i = 0; i < n1; i++) {
    printf("%d ", vec[i]);
  }
  printf("\n");
  printf("\n");
}

// TODO
__device__ void MAXIND(int k, int N, double *S, int *result) {
  int m = k + 1, i;
  for (i = k + 2; i < N; i++) {
    if (fabsf(S[INDEX(k, i, N, N)]) > fabsf(S[INDEX(k, m, N, N)])) {
      m = i;
    }
  }
  *result = m;
}

__device__ void UPDATE(int k, double t, double *e, bool *changed, int *state) {
  double ek_prev = e[k];
  e[k] = ek_prev + t;

  if (e[k] < 0) {
    e[k] = 0;
  }

  if (changed[k] && (ek_prev - e[k]) < JACOBI_TOLERANCE) {
    changed[k] = false;
    (*state)--;
  } else if ((!changed[k]) && (ek_prev - e[k]) > JACOBI_TOLERANCE) {
    changed[k] = true;
    (*state)++;
  }
}

__device__ void ROTATE(int k, int l, int i, int j, double c, double s,
                       double *S, int N) {
  double Skl = S[INDEX(k, l, N, N)], Sij = S[INDEX(i, j, N, N)];
  S[INDEX(k, l, N, N)] = c * Skl - s * Sij;
  S[INDEX(i, j, N, N)] = s * Skl + c * Sij;
}

__global__ void INIT1(int N, double *E) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N * N) {
    E[i] = ((i / N) == (i % N));
  }
}

__global__ void INIT2(int *state, int N) { *state = N; }

// TODO
__global__ void INIT3(int *ind, double *e, double *S, int N, bool *changed) {
  int k = blockIdx.x * blockDim.x + threadIdx.x;

  if (k < N) {
    MAXIND(k, N, S, &ind[k]);
    e[k] = S[INDEX(k, k, N, N)];
    changed[k] = true;
  }
}

// TODO
__global__ void BEST_M(int *m, int N, double *S, int *ind) {
  *m = 0;
  int k;
  for (k = 1; k < N - 1; k++) {
    if (fabs(S[INDEX(k, ind[k], N, N)]) > fabs(S[INDEX(*m, ind[*m], N, N)])) {
      *m = k;
    }
  }
}

__global__ void GET_S_C(int *k, int *l, int *m, double *c, double *s, double *t,
                        int N, int *ind, double *S, double *e) {
  *k = *m;
  *l = ind[*m];
  double p = S[INDEX(*k, *l, N, N)];
  double y = (e[*l] - e[*k]) / 2;
  double d = fabs(y) + sqrt(p * p + y * y);
  double r = sqrt(p * p + d * d);

  *c = d / r;
  *s = p / r;
  *t = p * p / d;

  if (y < 0) {
    *s = -(*s);
    *t = -(*t);
  }

  S[INDEX(*k, *l, N, N)] = 0.0;
}

__global__ void UPDATE_COMBINED(int *k, int *l, double *t, double *e,
                                bool *changed, int *state) {
  UPDATE(*k, -1 * (*t), e, changed, state);
  UPDATE(*l, *t, e, changed, state);
}

__global__ void ROTATE_MULTIPLE1(int *k, int *l, double *c, double *s,
                                 double *S, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < *k) {
    ROTATE(i, *k, i, *l, *c, *s, S, N);
  }
}

__global__ void ROTATE_MULTIPLE2(int *k, int *l, double *c, double *s,
                                 double *S, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + (*k) + 1;

  if (i < *l) {
    ROTATE(*k, i, i, *l, *c, *s, S, N);
  }
}

__global__ void ROTATE_MULTIPLE3(int *k, int *l, double *c, double *s,
                                 double *S, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + (*l) + 1;

  if (i < N) {
    ROTATE(*k, i, *l, i, *c, *s, S, N);
  }
}

__global__ void UPDATE_E(int N, double *E, int *k, int *l, double *c,
                         double *s) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  double Eik, Eil;

  if (i < N) {
    Eik = E[INDEX(i, *k, N, N)];
    Eil = E[INDEX(i, *l, N, N)];
    E[INDEX(i, *k, N, N)] = (*c) * Eik - (*s) * Eil;
    E[INDEX(i, *l, N, N)] = (*s) * Eik + (*c) * Eil;
  }
}

__global__ void UPDATE_IND(int *k, int *l, int *ind, int N, double *S) {
  MAXIND(*k, N, S, &ind[*k]);
  MAXIND(*l, N, S, &ind[*l]);
}

__global__ void TRANSPOSE(double *M, int m, int n, double *M_T) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n * m) {
    M_T[i] = M[INDEX(i % m, i / m, m, n)];
  }
}

__global__ void MATMUL2(int p, int q, int r, double *A, double *B, double *C) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int i;
  double sum = 0;

  if (row < p && col < r) {
    for (i = 0; i < q; i++) {
      sum += A[INDEX(row, i, p, q)] * B[INDEX(i, col, q, r)];
    }
    C[INDEX(row, col, p, r)] = sum;
  }
}

__global__ void ODD_EVEN_SORT(double *arr, int *indices, int n,
                              bool *converged) {

  int index_global = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  *converged = false;
  bool odd_iter = false;
  double temp;
  int to_see, to_see_next, index_local, i, temp_int;

  for (i = index_global; i < n; i += stride) {
    indices[i] = i;
  }

  while (!(*converged)) {
    *converged = true;
    for (index_local = index_global; index_local < n / 2;
         index_local += stride) {
      if (odd_iter && 2 * index_local + 2 < n) {
        to_see = 2 * index_local + 1;
        to_see_next = 2 * index_local + 2;
        if (arr[to_see] < arr[to_see_next]) {

          temp = arr[to_see_next];
          arr[to_see_next] = arr[to_see];
          arr[to_see] = temp;

          temp_int = indices[to_see_next];
          indices[to_see_next] = indices[to_see];
          indices[to_see] = temp_int;

          *converged = false;
        }
      } else if (!odd_iter && 2 * index_local + 1 < n) {
        to_see = 2 * index_local;
        to_see_next = 2 * index_local + 1;
        if (arr[to_see] < arr[to_see_next]) {

          temp = arr[to_see_next];
          arr[to_see_next] = arr[to_see];
          arr[to_see] = temp;

          temp_int = indices[to_see_next];
          indices[to_see_next] = indices[to_see];
          indices[to_see] = temp_int;

          *converged = false;
        }
      }
    }

    odd_iter = !odd_iter;
    __syncthreads();
  }
}

__global__ void ARRANGE(int *indices, double *old_E, double *new_E, int n1,
                        int n2) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n1 * n2; i += stride) {
    new_E[i] = old_E[INDEX(i / n2, indices[i % n2], n1, n2)];
  }
}

__global__ void GET_SINGULAR_VALS(int n, double *e, double *SIGMA,
                                  double *SIGMA_INV) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  int i;
  double sqrt_;

  for (i = index; i < n; i += stride) {
    sqrt_ = sqrt(e[i]);
    SIGMA[i] = sqrt_;
    SIGMA_INV[i] = 1 / sqrt_;
  }
}

// TODO
__global__ void GET_EIGEN_SUM(double *eigen_total, double *e, int n) {
  int i;
  *eigen_total = 0;
  for (i = 0; i < n; i++) {
    *eigen_total += e[i];
  }
}

__global__ void MULTIPLY_SIGMA_INV(int m, int n, double *M, double *V,
                                   double *SIGMA_INV, double *U) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int i;
  double sum = 0;

  if (row < m && col < m) {
    if (col < n) {
      for (i = 0; i < n; i++) {
        sum += M[INDEX(row, i, m, n)] * V[INDEX(i, col, n, n)];
      }
      U[INDEX(row, col, m, m)] = sum * SIGMA_INV[col];
    } else {
      U[INDEX(row, col, m, m)] = 0;
    }
  }
}

void GET_U(int m, int n, double *dev_M, double *dev_V, double *dev_SIGMA_INV,
           double *dev_U) {
  dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
  dim3 dimGrid((m + BLOCKSIZE - 1) / BLOCKSIZE,
               (m + BLOCKSIZE - 1) / BLOCKSIZE);
  MULTIPLY_SIGMA_INV<<<dimGrid, dimBlock>>>(m, n, dev_M, dev_V, dev_SIGMA_INV,
                                            dev_U);
}

__global__ void GET_RETENTION(int *k, int n, double *e, double *eigen_total,
                              double retention) {
  int k_retended = 0;
  double retention_done = 0;
  int i;

  for (i = 0; i < n; i++) {
    retention_done += 100 * e[i] / *eigen_total;
    k_retended++;
    if (retention_done >= retention) {
      break;
    }
  }

  *k = k_retended;
}

__global__ void GET_W(int k_retended, int n, double *W, double *E) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n * k_retended) {
    W[i] = E[INDEX(i / k_retended, i % k_retended, n, n)];
  }
}

void JACOBI(int n, double *dev_E, double *dev_e, double *dev_S) {

  int *dev_state, *dev_ind, *dev_m, *dev_k, *dev_l;
  double *dev_c, *dev_s, *dev_t_;
  bool *dev_changed;
  int state = n;

  hipMalloc(&dev_state, sizeof(int));
  hipMalloc(&dev_ind, sizeof(int) * n);
  hipMalloc(&dev_changed, sizeof(bool) * n);
  hipMalloc(&dev_m, sizeof(int));
  hipMalloc(&dev_k, sizeof(int));
  hipMalloc(&dev_l, sizeof(int));
  hipMalloc(&dev_c, sizeof(double));
  hipMalloc(&dev_s, sizeof(double));
  hipMalloc(&dev_t_, sizeof(double));

  int numblocks = (n * n + BLOCKSIZE - 1) / BLOCKSIZE;
  INIT1<<<numblocks, BLOCKSIZE>>>(n, dev_E);
  INIT2<<<1, 1>>>(dev_state, n);

  numblocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
  INIT3<<<numblocks, BLOCKSIZE>>>(dev_ind, dev_e, dev_S, n, dev_changed);

  int count = 0;

  while (state != 0 && count < 5 * n) {
    count++;

    // printf("%d %d\n", state, count);

    BEST_M<<<1, 1>>>(dev_m, n, dev_S, dev_ind);
    GET_S_C<<<1, 1>>>(dev_k, dev_l, dev_m, dev_c, dev_s, dev_t_, n, dev_ind,
                      dev_S, dev_e);
    UPDATE_COMBINED<<<1, 1>>>(dev_k, dev_l, dev_t_, dev_e, dev_changed,
                              dev_state);

    ROTATE_MULTIPLE1<<<numblocks, BLOCKSIZE>>>(dev_k, dev_l, dev_c, dev_s,
                                               dev_S, n);
    ROTATE_MULTIPLE2<<<numblocks, BLOCKSIZE>>>(dev_k, dev_l, dev_c, dev_s,
                                               dev_S, n);
    ROTATE_MULTIPLE3<<<numblocks, BLOCKSIZE>>>(dev_k, dev_l, dev_c, dev_s,
                                               dev_S, n);
    UPDATE_E<<<numblocks, BLOCKSIZE>>>(n, dev_E, dev_k, dev_l, dev_c, dev_s);
    UPDATE_IND<<<1, 1>>>(dev_k, dev_l, dev_ind, n, dev_S);

    hipMemcpy(&state, dev_state, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }

  hipFree(dev_state);
  hipFree(dev_ind);
  hipFree(dev_changed);
  hipFree(dev_m);
  hipFree(dev_k);
  hipFree(dev_l);
  hipFree(dev_c);
  hipFree(dev_s);
  hipFree(dev_t_);
}

void SVD_and_PCA(int m, int n, double *D, double **U, double **SIGMA,
                 double **V_T, int *SIGMAm, int *SIGMAn, double **D_HAT, int *K,
                 int retention) {

  double *dev_M, *dev_M_T, *dev_S, *dev_e, *dev_E, *dev_new_E, *dev_eigen_total,
      *dev_SIGMA, *dev_SIGMA_INV, *dev_V_T, *dev_U, *dev_W, *dev_D_HAT;

  int *dev_k, *dev_indices, numblocks = (m * n + BLOCKSIZE - 1) / BLOCKSIZE;

  hipMalloc(&dev_M, sizeof(double) * m * n);
  hipMemcpy(dev_M, D, sizeof(double) * m * n, hipMemcpyHostToDevice);
  hipMalloc(&dev_M_T, sizeof(double) * m * n);

  TRANSPOSE<<<numblocks, BLOCKSIZE>>>(dev_M, m, n, dev_M_T);

  hipMalloc(&dev_S, sizeof(double) * n * n);

  dim3 dimBlock(BLOCKSIZE, BLOCKSIZE);
  dim3 dimGrid((n + BLOCKSIZE - 1) / BLOCKSIZE,
               (n + BLOCKSIZE - 1) / BLOCKSIZE);

  MATMUL2<<<dimGrid, dimBlock>>>(n, m, n, dev_M_T, dev_M, dev_S);

  hipFree(dev_M_T);

  hipMalloc(&dev_e, sizeof(double) * n);
  hipMalloc(&dev_E, sizeof(double) * n * n);
  JACOBI(n, dev_E, dev_e, dev_S);

  hipFree(dev_S);

  hipMalloc(&dev_indices, sizeof(int) * n);
  hipMalloc(&dev_new_E, sizeof(double) * n * n);

  bool *converged;
  hipMalloc(&converged, sizeof(bool));
  numblocks = ((n / 2) + BLOCKSIZE - 1) / BLOCKSIZE;
  ODD_EVEN_SORT<<<numblocks, BLOCKSIZE>>>(dev_e, dev_indices, n, converged);
  hipFree(converged);

  numblocks = (n * n + BLOCKSIZE - 1) / BLOCKSIZE;
  ARRANGE<<<numblocks, BLOCKSIZE>>>(dev_indices, dev_E, dev_new_E, n, n);
  hipFree(dev_indices);

  hipFree(dev_E);
  dev_E = dev_new_E;

  hipMalloc(&dev_SIGMA, sizeof(double) * n);
  hipMalloc(&dev_SIGMA_INV, sizeof(double) * n);
  numblocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
  GET_SINGULAR_VALS<<<numblocks, BLOCKSIZE>>>(n, dev_e, dev_SIGMA,
                                              dev_SIGMA_INV);

  hipMalloc(&dev_eigen_total, sizeof(int));
  GET_EIGEN_SUM<<<1, 1>>>(dev_eigen_total, dev_e, n);

  hipMalloc(&dev_V_T, sizeof(double) * n * n);
  numblocks = (n * n + BLOCKSIZE - 1) / BLOCKSIZE;
  TRANSPOSE<<<numblocks, BLOCKSIZE>>>(dev_E, n, n, dev_V_T);

  hipMalloc(&dev_U, sizeof(double) * m * m);
  GET_U(m, n, dev_M, dev_E, dev_SIGMA_INV, dev_U);
  hipFree(dev_SIGMA_INV);

  hipMalloc(&dev_k, sizeof(int));
  GET_RETENTION<<<1, 1>>>(dev_k, n, dev_e, dev_eigen_total, retention);
  hipFree(dev_eigen_total);
  hipFree(dev_e);

  hipMemcpy(K, dev_k, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(dev_k);

  hipMalloc(&dev_W, sizeof(double) * n * (*K));
  hipMalloc(&dev_D_HAT, sizeof(double) * m * (*K));

  numblocks = (n * (*K) + BLOCKSIZE - 1) / BLOCKSIZE;
  GET_W<<<numblocks, BLOCKSIZE>>>(*K, n, dev_W, dev_E);

  hipFree(dev_E);

  dimGrid =
      dim3((*K + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
  MATMUL2<<<dimGrid, dimBlock>>>(m, n, *K, dev_M, dev_W, dev_D_HAT);

  hipFree(dev_W);
  hipFree(dev_M);

  *U = (double *)malloc(sizeof(double) * m * m);
  hipMemcpy(*U, dev_U, sizeof(double) * m * m, hipMemcpyDeviceToHost);
  hipFree(dev_U);

  *SIGMA = (double *)malloc(sizeof(double) * n);
  hipMemcpy(*SIGMA, dev_SIGMA, sizeof(double) * n, hipMemcpyDeviceToHost);
  hipFree(dev_SIGMA);

  *V_T = (double *)malloc(sizeof(double) * n * n);
  hipMemcpy(*V_T, dev_V_T, sizeof(double) * n * n, hipMemcpyDeviceToHost);
  hipFree(dev_V_T);

  *D_HAT = (double *)malloc(sizeof(double) * m * (*K));
  hipMemcpy(*D_HAT, dev_D_HAT, sizeof(double) * m * (*K),
             hipMemcpyDeviceToHost);

  // printMat<<<1, 1>>>(dev_U, m, m);

  hipFree(dev_D_HAT);

  hipDeviceSynchronize();

  *SIGMAm = m;
  *SIGMAn = n;
}
